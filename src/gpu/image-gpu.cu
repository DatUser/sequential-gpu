#include "hip/hip_runtime.h"
#include "image-gpu.hh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include <iostream>
#include <fstream>
#include <cstdio>

__global__
void to_gray_gpu(unsigned char* gray_data, unsigned char* data, int width, int height, int nb_channels);

ImageGPU::ImageGPU(const char* path) {
    unsigned char* stbi_data = stbi_load(path, &width, &height, &nb_channels, 0);
    if (!stbi_data) {
        std::cout << "FAILURE to load the image: " << path << '\n';
        return;
    }

    int size = width * height * nb_channels;

    // make allocations
    hipMallocManaged(&data, sizeof(unsigned char) * size);
    cudaCheckError();

    hipMallocManaged(&gray_data, sizeof(unsigned char) * width * height);
    cudaCheckError();

    // copy the data to GPU
    hipMemcpy(data, stbi_data, sizeof(unsigned char) * size, hipMemcpyHostToDevice);
    cudaCheckError();

    // free the stbi data
    stbi_image_free(stbi_data);
}

ImageGPU::~ImageGPU() {
    hipFree(data);
    hipFree(gray_data);
}

void ImageGPU::to_gray() {
    // TODO
    int nb_blocks_x = 50;
    int nb_blocks_y = 50;
    dim3 blocks_(nb_blocks_x, nb_blocks_y);
    dim3 threads_((height + nb_blocks_x) / nb_blocks_x, (width + nb_blocks_y) / nb_blocks_y);
    to_gray_gpu<<<blocks_, threads_>>>(gray_data, data, width, height, nb_channels);

    cudaCheckError();
    hipDeviceSynchronize();
    cudaCheckError();
}

void ImageGPU::save_gray_ppm(const char* path) const {
    std::ofstream ofs(path, std::ios_base::out | std::ios_base::binary);
    ofs << "P6" << std::endl << width << ' ' << height << std::endl << "255" << std::endl;

    for (int j = 0; j < height; ++j)
        for (int i = 0; i < width; ++i)
            ofs << (char) gray_data[j * width + i]
                << (char) gray_data[j * width + i]
                << (char) gray_data[j * width + i];

    ofs.close();
}

// -------------
// GPU functions
// -------------

__global__
void to_gray_gpu(unsigned char* gray_data, unsigned char* data, int width, int height, int nb_channels) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < height && j < width) {
        // get r / g / b
        float r = (float) data[i * width * nb_channels + j * nb_channels];
        float g = (float) data[i * width * nb_channels + j * nb_channels + 1];
        float b = (float) data[i * width * nb_channels + j * nb_channels + 2];

        // to gray
        float pixel_intensity = r * 0.2989 + g * 0.5870 + b * 0.1140;
        gray_data[i * width + j] = (unsigned char) pixel_intensity;
    }
}

