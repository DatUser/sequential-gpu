#include "blocks-gpu.hh"

#include "block-gpu.hh"

#include <iostream>

BlocksGPU::BlocksGPU(Blocks blocks, int w_size) {
    block_size = blocks.get_blocks()[0]->get_block_size();
    int size = block_size * block_size;
    nb_blocks = blocks.get_nb_blocks();
    window_size = w_size;

    hipMallocManaged(&textons_device, sizeof(unsigned char) * nb_blocks * size);
    cudaCheckError();

    hipMalloc(&blocks_device, sizeof(unsigned char) * nb_blocks * size);
    cudaCheckError();

    hipMallocManaged(&histogram, sizeof(int) * nb_blocks * size);
    cudaCheckError();

    for (int i = 0; i < nb_blocks; ++i) {
        // copy the data on the device
        hipMemcpy((blocks_device + i * size),
                   blocks.get_blocks()[i]->get_block(),
                   size * sizeof(unsigned char),
                   hipMemcpyHostToDevice);
        cudaCheckError();
    }
}

BlocksGPU::BlocksGPU(unsigned char* blocks_device,
                     int nb_blocks, int block_size,
                     int window_size) {
    int size = block_size * block_size;
    this->window_size = window_size;
    this->nb_blocks = nb_blocks;
    this->block_size = block_size;
    this->blocks_device = blocks_device;

    hipMallocManaged(&textons_device, sizeof(unsigned char) * nb_blocks * size); //TODO Added managed to debug
    cudaCheckError();

    //hipMalloc(&blocks_device, sizeof(unsigned char) * nb_blocks * size);
    //cudaCheckError();

    hipMallocManaged(&histogram, sizeof(int) * nb_blocks * size);
    cudaCheckError();

    hipMemset(histogram, 0, sizeof(int) * nb_blocks * size);
    cudaCheckError();
}


BlocksGPU::~BlocksGPU() {
    hipFree(textons_device);
    hipFree(blocks_device);
    hipFree(histogram);
}

void BlocksGPU::compute_textons() {
    int nb_blocks_cuda_x = 4;
    //std::cout << block_size << '\n';
    dim3 threads_(nb_blocks_cuda_x, block_size, block_size);
    dim3 blocks_((nb_blocks + nb_blocks_cuda_x) / nb_blocks_cuda_x, 1, 1);

    compute_texton_block_gpu<<<blocks_, threads_>>>(textons_device,
                                                    blocks_device,
                                                    block_size,
                                                    window_size,
                                                    nb_blocks);
    cudaCheckError();

    hipDeviceSynchronize();
    cudaCheckError();
}

void BlocksGPU::compute_histogram_blocks() {
    int size = block_size * block_size;
    //for (int i = 0; i < nb_blocks; ++i) {
    dim3 threads_(size);
    dim3 blocks_(nb_blocks);

    compute_histogram_block_gpu<<<blocks_, threads_>>>(histogram /*+ i * size*/
    , textons_device /*+ i * size*/, size, nb_blocks);

    cudaCheckError();
    hipDeviceSynchronize();
    cudaCheckError();
    //}

}

void BlocksGPU::compute_shared_histogram_blocks() {
    int size = block_size * block_size;
    dim3 threads_(size);
    dim3 blocks_(nb_blocks);

    compute_shared_histogram_block_gpu<<<blocks_, threads_, size * sizeof(int)>>>(histogram, textons_device, size, nb_blocks);

    cudaCheckError();
    hipDeviceSynchronize();
    cudaCheckError();
}
